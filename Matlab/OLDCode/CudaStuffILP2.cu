#include "hip/hip_runtime.h"
#include "CudaStuff.cuh"
#include "Util.h"

XXX #define WARPSIZE 32

XXX For BeforeLU no change at all - all is predone is MATLAB. Just change to P32.
__device__ void BeforeLU(HMat InMat, MYFTYPE* B, MYFTYPE* uHP, MYFTYPE* bHP)
{
	MYDTYPE JumctionI =blockIdx.x*blockDim.x + threadIdx.x,i=0,j=0,CurJ,CurB,t, CurLevel;
	CurB = JumctionI;
	uHP[JumctionI]=InMat.d[JumctionI];
	bHP[JumctionI]=B[JumctionI];
		for(CurLevel=0;CurLevel<=InMat.Depth;CurLevel++) {
			if(InMat.Level[JumctionI]==CurLevel) {
				for(i=InMat.SegStartI[JumctionI]-1;i<InMat.SegEndI[JumctionI];i++) {
					MYFTYPE uHPm1=uHP[i-1];
					uHP[i]=uHP[i]-InMat.e[i]*(InMat.f[i-1]/uHPm1); // So far same as paper parallel
					uHPm1=uHP[i-1];
					MYFTYPE bHPm1=bHP[i-1];
					bHP[i]=bHP[i]-bHPm1*InMat.e[i]/uHPm1; // bH is y
				}
			}	
			__syncthreads();
			if(InMat.Level[CurB]==(CurLevel+1)) {
				CurJ=InMat.BranchP[CurB-1]-1;
				MYDTYPE St=InMat.RelStarts[CurB];
				MYDTYPE En=InMat.RelEnds[CurB];
				for(j=St;j<=En;j++) {
					t=InMat.RelVec[j-1]-1;
					MYFTYPE uHPm1=uHP[t-1];
					uHP[CurJ]=uHP[CurJ]-InMat.e[t]*(InMat.f[t-1]/uHPm1); 
					uHPm1=uHP[t-1];
					MYFTYPE bHPm1=bHP[t-1];
					bHP[CurJ]=bHP[CurJ]-bHPm1*InMat.e[t]/uHPm1; 
				}
			}	
			__syncthreads();
		}
	}
}

__device__ void BkSub(HMat InMat, MYFTYPE* PX, MYFTYPE* PF,MYFTYPE* uHP, MYFTYPE* bHP)
{
	MYDTYPE NextID,i,j;
	XXX MYDTYPE NextID_2,j_2;
	j =blockIdx.x*blockDim.x + threadIdx.x;
	XXX j_2 = j+WARPSIZE;
	PX[j]=bHP[j]/uHP[j];
	XXX PX[j_2]=bHP[j_2]/uHP[j_2];
	PF[j]=-InMat.f[j]/uHP[j];
	XXX PF[j_2]=-InMat.f[j_2]/uHP[j_2];
	PX[InMat.N]=0;
	PF[InMat.N]=1;
	for(i=0;i<InMat.LognDepth;i++) {	
		NextID=InMat.FIdxs[i+j*(InMat.LognDepth)]-1;
		XXX NextID_2=InMat.FIdxs[i+j_2*(InMat.LognDepth)]-1;
		MYFTYPE OldPXj=PX[j];
		XXX MYFTYPE OldPXj_2=PX[j_2];
		MYFTYPE OldPXNextID=PX[NextID];
		XXX MYFTYPE OldPXNextID_2=PX[NextID_2];
		PX[j]=OldPXj+OldPXNextID*PF[j];
		XXX PX[j_2]=OldPXj_2+OldPXNextID_2*PF[j_2];
		MYFTYPE OldPFj=PF[j];
		XXX MYFTYPE OldPFj_2=PF[j_2];
		MYFTYPE OldPFNextID=PF[NextID];
		XXX MYFTYPE OldPFNextID_2=PF[NextID_2];
		PF[j]=OldPFj*OldPFNextID;
		XXX PF[j_2]=OldPFj_2*OldPFNextID_2;
	}
}

__global__ void stEfork2TimeLoopGPUKernel(Stim stim, const HHparams InHHParams, HMat InMat, MYDTYPE Nt, MYFTYPE dt, MYDTYPE N, MYFTYPE *Iapp, MYFTYPE stimArea, MYFTYPE Cm, MYFTYPE *VHot, MYFTYPE *V, MYFTYPE *n, MYFTYPE *m, MYFTYPE *h, MYFTYPE *dOrig, MYFTYPE *d)
{
	MYDTYPE i=blockIdx.x*blockDim.x + threadIdx.x;
	MYDTYPE i_2=i+WARPSIZE;
	
	InMat.d=d;

	// for model
	MYFTYPE t;
	XXX MYFTYPE a_2, c_2, n4_2, m3h_2, TmpVec_2, TmpVec2_2, dVec_2, Vmid_2;
	MYFTYPE a, c, n4, m3h, TmpVec, TmpVec2, dVec, Vmid;
	XXX MYFTYPE v_2=V[i_2];
	MYFTYPE v=V[i];

	// For solving the matrix
	MYFTYPE *B=(MYFTYPE*) &smem[0];
	MYFTYPE *uHP,*bHP,*PX,*PF;
	uHP = (MYFTYPE*) &smem[InMat.N];
	bHP = (MYFTYPE*) &smem[2*InMat.N];
	//try to use only 2*N space  in shared memmo
	PX = (MYFTYPE*) &smem[3*InMat.N];
	PF = (MYFTYPE*) &smem[4*InMat.N+1];

	// Optimization - Mat on Shared memory
	MYFTYPE *SMemd,*SMeme,*SMemf,*SMemVHot;
	MYFTYPE *Oldd,*Olde,*Oldf,*OldVHot;
	Oldd=InMat.d;
	Olde=InMat.e;
	Oldf=InMat.f;
	OldVHot=VHot;
	SMemd = (MYFTYPE*) &smem[5*InMat.N+3];
	SMeme = (MYFTYPE*) &smem[6*InMat.N+3];
	SMemf = (MYFTYPE*) &smem[7*InMat.N+3];
	SMemVHot = (MYFTYPE*) &smem[8*InMat.N+3];
	SMemd[i]=InMat.d[i];
	XXX SMemd[i_2]=InMat.d[i_2];
	SMeme[i]=InMat.e[i];
	XXX 
	SMemf[i]=InMat.f[i];
	XXX 
	SMemVHot[0]=VHot[0];
	InMat.d=SMemd;
	InMat.e=SMeme;
	InMat.f=SMemf;
	d=InMat.d;
	VHot=SMemVHot;
	/* Call to solve matrix -
	BeforeLU(InMat,B,uHP,bHP);
	__syncthreads();
	BkSub(InMat, PX,PF,uHP,bHP);*/

	__syncthreads();
	for(int j=1;j<Nt;j++) {
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();

		t = j*dt;
		// I = Iapp*(t-dt/2>stim.t1)*(t-dt/2<stim.t2);
		if(t>stim.t1+dt/2 && t<stim.t2+dt/2) { 
			Iapp[stim.loc] = stim.amp/stimArea; } // All together?
		else {
			Iapp[stim.loc] = 0; // All together?
		}
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();// a = an(V);  c = (a+bn(V))/2;
		// cu_an(V[i],a);
		a =.01*(10-(v+71))/(exp(1-(v+71)/10)-1);
		XXX a_2 =.01*(10-(v_2+71))/(exp(1-(v_2+71)/10)-1);
		// cu_bn(V[i],c);
		c = .125*exp(-((v+71)/80));
		XXX c_2 = .125*exp(-((v_2+71)/80));
		c+=a;
		XXX c_2+=a_2;
		c*=0.5f;
		XXX c_2*=0.5f;

		//  n = ( (1/dt-c).*n + a) ./ (1/dt + c); n4 = n.^4;
		// Parallel ! for(int i=0;i<N;i++) {
		n[i]=( (1/dt-c)*n[i] + a) / (1/dt + c);
		XXX n[i_2]=( (1/dt-c_2)*n[i_2] + a_2) / (1/dt + c_2);
		n4=pow(n[i],4);
		XXX n4_2=pow(n[i_2],4);
		// }
		// a = am(V);  c = (a+bm(V))/2;
		// cu_am(V[i],a);
		a = .1*(25-(v+71))/(exp(2.5-(v+71)/10)-1);
		XXX 
		// cu_bm(V[i],c);
		c = 4*exp(-(v+71)/18);
		XXX 
		c+=a;
		XXX 
		c*=0.5f;
		XXX 
		// m = ( (1/dt-c).*m + a) ./ (1/dt + c);
		// Parallel ! for(int i=0;i<N;i++) {
		m[i]=( (1/dt-c)*m[i] + a) / (1/dt + c);
		// }
		// a = ah(V);  c = (a+bh(V))/2;
		// cu_ah(V[i],a);
		a=0.07*exp(-(v+71)/20);
		// cu_bh(V[i],c);
		c=1./(exp(3-(v+71)/10)+1);
		c+=a;
		c*=0.5f;
		// h = ( (1/dt-c).*h + a) ./ (1/dt + c); m3h = m.^3.*h;
		// Parallel ! for(int i=0;i<N;i++) {
		h[i]=( (1/dt-c)*h[i] + a) / (1/dt + c);
		m3h=pow(m[i],3)*h[i];
		// }
		// d = g.Na.*m3h + g.K.*n4 + g.Cl;
		/*TmpVec=m3h;
		TmpVec*=InHHParams.gNa;
		TmpVec2=n4;
		TmpVec2*=InHHParams.gK;
		TmpVec+=TmpVec2;
		TmpVec+=InHHParams.gCl;
		dVec=TmpVec;*/
		// or
		dVec=m3h*InHHParams.gNa+n4*InHHParams.gK+InHHParams.gCl;
		XXX dVec_2=m3h_2*InHHParams.gNa+n4_2*InHHParams.gK+InHHParams.gCl;

		// f = g.Na.*m3h*E.Na + g.K.*n4*E.K + g.Cl.*E.Cl + I;
		/*cu_CopyVec(TmpVec,m3h,N);
		cu_VecMultScalar(TmpVec,InHHParams.gNa*InHHParams.ENa,N);
		cu_CopyVec(TmpVec2,n4,N);
		cu_VecMultScalar(TmpVec2,InHHParams.gK*InHHParams.EK,N);
		cu_VecPlusVec(TmpVec,TmpVec2,N);
		cu_VecPlusScalar(TmpVec,InHHParams.gCl*InHHParams.ECl,N);
		cu_VecPlusVec(TmpVec,Iapp,N); // TmpVec is f!*/
		TmpVec=m3h*InHHParams.gNa*InHHParams.ENa+n4*InHHParams.gK*InHHParams.EK+InHHParams.gCl*InHHParams.ECl+Iapp[i];
		XXX 
		
		// B(1:Nx+1:end) = dB + d + 2*Cm/dt;         % update the diagonal
		/*cu_CopyVec(d,dOrig,N);
		cu_VecPlusVec(d,dVec,N);
		cu_VecPlusScalar(d,2*Cm/dt,N);*/
		d[i]=dOrig[i]+dVec+2*Cm/dt;
		XXX d[i_2]=dOrig[i_2]+dVec_2+2*Cm/dt;
		
		//Vmid = B\(2*Cm*V/dt + f); % SOLUTION OF MATRIX
		// Mat=B;
		// Vec=2*Cm*V/dt + f;
		// %       Vmid = SolveNormally(Mat,Vec);
		/*cu_CopyVec(TmpVec2,V,N);
		cu_VecMultScalar(TmpVec2,2*Cm/dt,N);
		cu_VecPlusVec(TmpVec2,TmpVec,N);*/
		B[i]=v*2*Cm/dt+TmpVec;
		XXX B[i_2]=v_2*2*Cm/dt+TmpVec_2;
		// Vmid2 = SolveByUs(Mat,Vec,Aux)';

		//SolveTriDiagonalHinesSerialCPU(InMat, TmpVec2, Vmid);

		// SolveTriDiagonalGPU(InMat, TmpVec2, Vmid);
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		// Call to solve matrix -
		BeforeLU(InMat,B,uHP,bHP);
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		BkSub(InMat, PX,PF,uHP,bHP);
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		Vmid=PX[i];
		XXX Vmid_2=PX[i_2];
		// End Call to solve matrix

		//cudaStatus = hipDeviceSynchronize();

		// Err(j)=sum(abs(Vmid-Vmid2));
		// V = 2*Vmid - V;
		/* cu_VecMultScalar(V,-1,N);
		cu_VecMultScalar(Vmid,2,N);
		cu_VecPlusVec(V,Vmid,N); */
		v=Vmid*2-v;
		XXX v_2=Vmid_2*2-v_2;
		V[i]=v;
		V[i_2]=v_2;
		if(i==stim.loc) {
			VHot[j] = V[stim.loc]; } // All together?
	}

	// return data to global memory
	Oldd[i]=SMemd[i];
	XXX Oldd[i_2]=SMemd[i_2];
	Olde[i]=SMeme[i];
	XXX 
	Oldf[i]=SMemf[i];
	XXX 
	InMat.d=Oldd;
	InMat.e=Olde;
	InMat.f=Oldf;

	XXX XXX change XXX for(int j=0;j<Nt/N;j++) {
		OldVHot[i+j*N]=VHot[i+j*N];
	}
	VHot=OldVHot;
}