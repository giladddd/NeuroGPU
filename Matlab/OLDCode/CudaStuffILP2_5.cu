#include "hip/hip_runtime.h"
#include "CudaStuff.cuh"
#include "Util.h"

XXX #define WARPSIZE 32

XXX For BeforeLU no change at all - all is predone is MATLAB. Just change to P32.
__device__ void BeforeLU(HMat InMat, MYFTYPE* B, MYFTYPE* uHP, MYFTYPE* bHP)
{
	MYDTYPE JumctionI =blockIdx.x*blockDim.x + threadIdx.x,i=0,j=0,CurJ,CurB,t, CurLevel;
	CurB = JumctionI;
	uHP[JumctionI]=InMat.d[JumctionI];
	bHP[JumctionI]=B[JumctionI];
		for(CurLevel=0;CurLevel<=InMat.Depth;CurLevel++) {
			if(InMat.Level[JumctionI]==CurLevel) {
				for(i=InMat.SegStartI[JumctionI]-1;i<InMat.SegEndI[JumctionI];i++) {
					MYFTYPE uHPm1=uHP[i-1];
					uHP[i]=uHP[i]-InMat.e[i]*(InMat.f[i-1]/uHPm1); // So far same as paper parallel
					uHPm1=uHP[i-1];
					MYFTYPE bHPm1=bHP[i-1];
					bHP[i]=bHP[i]-bHPm1*InMat.e[i]/uHPm1; // bH is y
				}
			}	
			__syncthreads();
			if(InMat.Level[CurB]==(CurLevel+1)) {
				CurJ=InMat.BranchP[CurB-1]-1;
				MYDTYPE St=InMat.RelStarts[CurB];
				MYDTYPE En=InMat.RelEnds[CurB];
				for(j=St;j<=En;j++) {
					t=InMat.RelVec[j-1]-1;
					MYFTYPE uHPm1=uHP[t-1];
					uHP[CurJ]=uHP[CurJ]-InMat.e[t]*(InMat.f[t-1]/uHPm1); 
					uHPm1=uHP[t-1];
					MYFTYPE bHPm1=bHP[t-1];
					bHP[CurJ]=bHP[CurJ]-bHPm1*InMat.e[t]/uHPm1; 
				}
			}	
			__syncthreads();
		}
	}
}

__device__ void BkSub(HMat InMat, MYFTYPE* PX, MYFTYPE* PF,MYFTYPE* uHP, MYFTYPE* bHP)
{
	MYDTYPE NextID,i,j;
	XXX MYDTYPE NextID_2,j_2;
	XXX MYDTYPE NextID_3,j_3;
	XXX MYDTYPE NextID_4,j_4;
	XXX MYDTYPE NextID_5,j_5;
	j =blockIdx.x*blockDim.x + threadIdx.x;
	XXX j_2 = j+WARPSIZE;
	XXX j_3 = j+(WARPSIZE*2);
	XXX j_4 = j+(WARPSIZE*3);
	XXX j_5 = j+(WARPSIZE*4);
	PX[j]=bHP[j]/uHP[j];
	XXX PX[j_2]=bHP[j_2]/uHP[j_2];
	XXX PX[j_3]=bHP[j_3]/uHP[j_3];
	XXX PX[j_4]=bHP[j_4]/uHP[j_4];
	XXX PX[j_5]=bHP[j_5]/uHP[j_5];
	PF[j]=-InMat.f[j]/uHP[j];
	XXX PF[j_2]=-InMat.f[j_2]/uHP[j_2];
	XXX PF[j_3]=-InMat.f[j_3]/uHP[j_3];
	XXX PF[j_4]=-InMat.f[j_4]/uHP[j_4];
	XXX PF[j_5]=-InMat.f[j_5]/uHP[j_5];
	PX[InMat.N]=0;
	PF[InMat.N]=1;
	for(i=0;i<InMat.LognDepth;i++) {	
		NextID=InMat.FIdxs[i+j*(InMat.LognDepth)]-1;
		XXX NextID_2=InMat.FIdxs[i+j_2*(InMat.LognDepth)]-1;
		XXX NextID_3=InMat.FIdxs[i+j_3*(InMat.LognDepth)]-1;
		XXX NextID_4=InMat.FIdxs[i+j_4*(InMat.LognDepth)]-1;
		XXX NextID_5=InMat.FIdxs[i+j_5*(InMat.LognDepth)]-1;
		MYFTYPE OldPXj=PX[j];
		XXX MYFTYPE OldPXj_2=PX[j_2];
		XXX MYFTYPE OldPXj_3=PX[j_3];
		XXX MYFTYPE OldPXj_4=PX[j_4];
		XXX MYFTYPE OldPXj_5=PX[j_5];
		MYFTYPE OldPXNextID=PX[NextID];
		XXX MYFTYPE OldPXNextID_2=PX[NextID_2];
		XXX MYFTYPE OldPXNextID_3=PX[NextID_3];
		XXX MYFTYPE OldPXNextID_4=PX[NextID_4];
		XXX MYFTYPE OldPXNextID_5=PX[NextID_5];
		PX[j]=OldPXj+OldPXNextID*PF[j];
		XXX PX[j_2]=OldPXj_2+OldPXNextID_2*PF[j_2];
		XXX PX[j_3]=OldPXj_3+OldPXNextID_3*PF[j_3];
		XXX PX[j_4]=OldPXj_4+OldPXNextID_4*PF[j_4];
		XXX PX[j_5]=OldPXj_5+OldPXNextID_5*PF[j_5];
		MYFTYPE OldPFj=PF[j];
		XXX MYFTYPE OldPFj_2=PF[j_2];
		XXX MYFTYPE OldPFj_3=PF[j_3];
		XXX MYFTYPE OldPFj_4=PF[j_4];
		XXX MYFTYPE OldPFj_5=PF[j_5];
		MYFTYPE OldPFNextID=PF[NextID];
		XXX MYFTYPE OldPFNextID_2=PF[NextID_2];
		XXX MYFTYPE OldPFNextID_3=PF[NextID_3];
		XXX MYFTYPE OldPFNextID_4=PF[NextID_4];
		XXX MYFTYPE OldPFNextID_5=PF[NextID_5];
		PF[j]=OldPFj*OldPFNextID;
		XXX PF[j_2]=OldPFj_2*OldPFNextID_2;
		XXX PF[j_3]=OldPFj_3*OldPFNextID_3;
		XXX PF[j_4]=OldPFj_4*OldPFNextID_4;
		XXX PF[j_5]=OldPFj_5*OldPFNextID_5;
	}
}

__global__ void stEfork2TimeLoopGPUKernel(Stim stim, const HHparams InHHParams, HMat InMat, MYDTYPE Nt, MYFTYPE dt, MYDTYPE N, MYFTYPE *Iapp, MYFTYPE stimArea, MYFTYPE Cm, MYFTYPE *VHot, MYFTYPE *V, MYFTYPE *n, MYFTYPE *m, MYFTYPE *h, MYFTYPE *dOrig, MYFTYPE *d)
{
	MYDTYPE i=blockIdx.x*blockDim.x + threadIdx.x;
	MYDTYPE i_2=i+WARPSIZE;
	MYDTYPE i_3=i+(WARPSIZE*2);
	MYDTYPE i_4=i+(WARPSIZE*3);
	MYDTYPE i_5=i+(WARPSIZE*4);
	
	InMat.d=d;

	// for model
	MYFTYPE t;
	XXX MYFTYPE a_2, c_2, n4_2, m3h_2, TmpVec_2, TmpVec2_2, dVec_2, Vmid_2;
	XXX MYFTYPE a_3, c_3, n4_3, m3h_3, TmpVec_3, TmpVec2_3, dVec_3, Vmid_3;
	XXX MYFTYPE a_4, c_4, n4_4, m3h_4, TmpVec_4, TmpVec2_4, dVec_4, Vmid_4;
	XXX MYFTYPE a_5, c_5, n4_5, m3h_5, TmpVec_5, TmpVec2_5, dVec_5, Vmid_5;
	MYFTYPE a, c, n4, m3h, TmpVec, TmpVec2, dVec, Vmid;
	XXX MYFTYPE v_2=V[i_2];
	XXX MYFTYPE v_3=V[i_3];
	XXX MYFTYPE v_4=V[i_4];
	XXX MYFTYPE v_5=V[i_5];
	MYFTYPE v=V[i];

	// For solving the matrix
	MYFTYPE *B=(MYFTYPE*) &smem[0];
	MYFTYPE *uHP,*bHP,*PX,*PF;
	uHP = (MYFTYPE*) &smem[InMat.N];
	bHP = (MYFTYPE*) &smem[2*InMat.N];
	//try to use only 2*N space  in shared memmo
	PX = (MYFTYPE*) &smem[3*InMat.N];
	PF = (MYFTYPE*) &smem[4*InMat.N+1];

	// Optimization - Mat on Shared memory
	MYFTYPE *SMemd,*SMeme,*SMemf,*SMemVHot;
	MYFTYPE *Oldd,*Olde,*Oldf,*OldVHot;
	Oldd=InMat.d;
	Olde=InMat.e;
	Oldf=InMat.f;
	OldVHot=VHot;
	SMemd = (MYFTYPE*) &smem[5*InMat.N+3];
	SMeme = (MYFTYPE*) &smem[6*InMat.N+3];
	SMemf = (MYFTYPE*) &smem[7*InMat.N+3];
	SMemVHot = (MYFTYPE*) &smem[8*InMat.N+3];
	SMemd[i]=InMat.d[i];
	XXX SMemd[i_2]=InMat.d[i_2];
	XXX SMemd[i_3]=InMat.d[i_3];
	XXX SMemd[i_4]=InMat.d[i_4];
	XXX SMemd[i_5]=InMat.d[i_5];
	SMeme[i]=InMat.e[i];
	XXX 
	SMemf[i]=InMat.f[i];
	XXX 
	SMemVHot[0]=VHot[0];
	InMat.d=SMemd;
	InMat.e=SMeme;
	InMat.f=SMemf;
	d=InMat.d;
	VHot=SMemVHot;
	/* Call to solve matrix -
	BeforeLU(InMat,B,uHP,bHP);
	__syncthreads();
	BkSub(InMat, PX,PF,uHP,bHP);*/

	__syncthreads();
	for(int j=1;j<Nt;j++) {
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();

		t = j*dt;
		// I = Iapp*(t-dt/2>stim.t1)*(t-dt/2<stim.t2);
		if(t>stim.t1+dt/2 && t<stim.t2+dt/2) { 
			Iapp[stim.loc] = stim.amp/stimArea; } // All together?
		else {
			Iapp[stim.loc] = 0; // All together?
		}
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();// a = an(V);  c = (a+bn(V))/2;
		// cu_an(V[i],a);
		a =.01*(10-(v+71))/(exp(1-(v+71)/10)-1);
		XXX a_2 =.01*(10-(v_2+71))/(exp(1-(v_2+71)/10)-1);
		XXX a_3 =.01*(10-(v_3+71))/(exp(1-(v_3+71)/10)-1);
		XXX a_4 =.01*(10-(v_4+71))/(exp(1-(v_4+71)/10)-1);
		XXX a_5 =.01*(10-(v_5+71))/(exp(1-(v_5+71)/10)-1);
		// cu_bn(V[i],c);
		c = .125*exp(-((v+71)/80));
		XXX c_2 = .125*exp(-((v_2+71)/80));
		XXX c_3 = .125*exp(-((v_3+71)/80));
		XXX c_4 = .125*exp(-((v_4+71)/80));
		XXX c_5 = .125*exp(-((v_5+71)/80));
		c+=a;
		XXX c_2+=a_2;
		XXX c_3+=a_3;
		XXX c_4+=a_4;
		XXX c_5+=a_5;
		c*=0.5f;
		XXX c_2*=0.5f;
		XXX c_3*=0.5f;
		XXX c_4*=0.5f;
		XXX c_5*=0.5f;

		//  n = ( (1/dt-c).*n + a) ./ (1/dt + c); n4 = n.^4;
		// Parallel ! for(int i=0;i<N;i++) {
		n[i]=( (1/dt-c)*n[i] + a) / (1/dt + c);
		XXX n[i_2]=( (1/dt-c_2)*n[i_2] + a_2) / (1/dt + c_2);
		XXX n[i_3]=( (1/dt-c_3)*n[i_3] + a_3) / (1/dt + c_3);
		XXX n[i_4]=( (1/dt-c_4)*n[i_4] + a_4) / (1/dt + c_4);
		XXX n[i_5]=( (1/dt-c_5)*n[i_5] + a_5) / (1/dt + c_5);
		n4=pow(n[i],4);
		XXX n4_2=pow(n[i_2],4);
		XXX n4_3=pow(n[i_3],4);
		XXX n4_4=pow(n[i_4],4);
		XXX n4_5=pow(n[i_5],4);
		// }
		// a = am(V);  c = (a+bm(V))/2;
		// cu_am(V[i],a);
		a = .1*(25-(v+71))/(exp(2.5-(v+71)/10)-1);
		XXX 
		// cu_bm(V[i],c);
		c = 4*exp(-(v+71)/18);
		XXX 
		c+=a;
		XXX 
		c*=0.5f;
		XXX 
		// m = ( (1/dt-c).*m + a) ./ (1/dt + c);
		// Parallel ! for(int i=0;i<N;i++) {
		m[i]=( (1/dt-c)*m[i] + a) / (1/dt + c);
		// }
		// a = ah(V);  c = (a+bh(V))/2;
		// cu_ah(V[i],a);
		a=0.07*exp(-(v+71)/20);
		// cu_bh(V[i],c);
		c=1./(exp(3-(v+71)/10)+1);
		c+=a;
		c*=0.5f;
		// h = ( (1/dt-c).*h + a) ./ (1/dt + c); m3h = m.^3.*h;
		// Parallel ! for(int i=0;i<N;i++) {
		h[i]=( (1/dt-c)*h[i] + a) / (1/dt + c);
		m3h=pow(m[i],3)*h[i];
		// }
		// d = g.Na.*m3h + g.K.*n4 + g.Cl;
		/*TmpVec=m3h;
		TmpVec*=InHHParams.gNa;
		TmpVec2=n4;
		TmpVec2*=InHHParams.gK;
		TmpVec+=TmpVec2;
		TmpVec+=InHHParams.gCl;
		dVec=TmpVec;*/
		// or
		dVec=m3h*InHHParams.gNa+n4*InHHParams.gK+InHHParams.gCl;
		XXX dVec_2=m3h_2*InHHParams.gNa+n4_2*InHHParams.gK+InHHParams.gCl;
		XXX dVec_3=m3h_3*InHHParams.gNa+n4_3*InHHParams.gK+InHHParams.gCl;
		XXX dVec_4=m3h_4*InHHParams.gNa+n4_4*InHHParams.gK+InHHParams.gCl;
		XXX dVec_5=m3h_5*InHHParams.gNa+n4_5*InHHParams.gK+InHHParams.gCl;

		// f = g.Na.*m3h*E.Na + g.K.*n4*E.K + g.Cl.*E.Cl + I;
		/*cu_CopyVec(TmpVec,m3h,N);
		cu_VecMultScalar(TmpVec,InHHParams.gNa*InHHParams.ENa,N);
		cu_CopyVec(TmpVec2,n4,N);
		cu_VecMultScalar(TmpVec2,InHHParams.gK*InHHParams.EK,N);
		cu_VecPlusVec(TmpVec,TmpVec2,N);
		cu_VecPlusScalar(TmpVec,InHHParams.gCl*InHHParams.ECl,N);
		cu_VecPlusVec(TmpVec,Iapp,N); // TmpVec is f!*/
		TmpVec=m3h*InHHParams.gNa*InHHParams.ENa+n4*InHHParams.gK*InHHParams.EK+InHHParams.gCl*InHHParams.ECl+Iapp[i];
		XXX 
		
		// B(1:Nx+1:end) = dB + d + 2*Cm/dt;         % update the diagonal
		/*cu_CopyVec(d,dOrig,N);
		cu_VecPlusVec(d,dVec,N);
		cu_VecPlusScalar(d,2*Cm/dt,N);*/
		d[i]=dOrig[i]+dVec+2*Cm/dt;
		XXX d[i_2]=dOrig[i_2]+dVec_2+2*Cm/dt;
		XXX d[i_3]=dOrig[i_3]+dVec_3+2*Cm/dt;
		XXX d[i_4]=dOrig[i_4]+dVec_4+2*Cm/dt;
		XXX d[i_5]=dOrig[i_5]+dVec_5+2*Cm/dt;
		
		//Vmid = B\(2*Cm*V/dt + f); % SOLUTION OF MATRIX
		// Mat=B;
		// Vec=2*Cm*V/dt + f;
		// %       Vmid = SolveNormally(Mat,Vec);
		/*cu_CopyVec(TmpVec2,V,N);
		cu_VecMultScalar(TmpVec2,2*Cm/dt,N);
		cu_VecPlusVec(TmpVec2,TmpVec,N);*/
		B[i]=v*2*Cm/dt+TmpVec;
		XXX B[i_2]=v_2*2*Cm/dt+TmpVec_2;
		XXX B[i_3]=v_3*2*Cm/dt+TmpVec_3;
		XXX B[i_4]=v_4*2*Cm/dt+TmpVec_4;
		XXX B[i_5]=v_5*2*Cm/dt+TmpVec_5;
		// Vmid2 = SolveByUs(Mat,Vec,Aux)';

		//SolveTriDiagonalHinesSerialCPU(InMat, TmpVec2, Vmid);

		// SolveTriDiagonalGPU(InMat, TmpVec2, Vmid);
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		// Call to solve matrix -
		BeforeLU(InMat,B,uHP,bHP);
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		BkSub(InMat, PX,PF,uHP,bHP);
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		__syncthreads();
		Vmid=PX[i];
		XXX Vmid_2=PX[i_2];
		XXX Vmid_3=PX[i_3];
		XXX Vmid_4=PX[i_4];
		XXX Vmid_5=PX[i_5];
		// End Call to solve matrix

		//cudaStatus = hipDeviceSynchronize();

		// Err(j)=sum(abs(Vmid-Vmid2));
		// V = 2*Vmid - V;
		/* cu_VecMultScalar(V,-1,N);
		cu_VecMultScalar(Vmid,2,N);
		cu_VecPlusVec(V,Vmid,N); */
		v=Vmid*2-v;
		XXX v_2=Vmid_2*2-v_2;
		XXX v_3=Vmid_3*2-v_3;
		XXX v_4=Vmid_4*2-v_4;
		XXX v_5=Vmid_5*2-v_5;
		V[i]=v;
		V[i_2]=v_2;
		V[i_3]=v_3;
		V[i_4]=v_4;
		V[i_5]=v_5;
		if(i==stim.loc) {
			VHot[j] = V[stim.loc]; } // All together?
	}

	// return data to global memory
	Oldd[i]=SMemd[i];
	XXX Oldd[i_2]=SMemd[i_2];
	XXX Oldd[i_3]=SMemd[i_3];
	XXX Oldd[i_4]=SMemd[i_4];
	XXX Oldd[i_5]=SMemd[i_5];
	Olde[i]=SMeme[i];
	XXX 
	Oldf[i]=SMemf[i];
	XXX 
	InMat.d=Oldd;
	InMat.e=Olde;
	InMat.f=Oldf;

	XXX XXX change XXX for(int j=0;j<Nt/N;j++) {
		OldVHot[i+j*N]=VHot[i+j*N];
	}
	VHot=OldVHot;
}
