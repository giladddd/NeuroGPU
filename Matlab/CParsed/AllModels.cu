#include "hip/hip_runtime.h"
// Automatically generated CU for C:\Users\bensr\Documents\GitHub\NeuroGPU\URapNeuron\HodgkinHuxley\runModel.hoc
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""
#include "AllModels.cuh"

// Universals:
#define PI (3.1415927f)
#define R (8.31441f)
#define FARADAY (96485.309f)
#define ktf (1000.*8.3134*(celsius + 273.15)/FARADAY)

// GGlobals
#define celsius (6.30000)
#define stoprun (0.00000)
#define clamp_resist (0.00100)
#define secondorder (0.00000)

// NGlobals:
#define minf_hh2 (0.052932)
#define hinf_hh2 (0.59612)
#define ninf_hh2 (0.31768)
#define mtau_hh2 (0.23677)
#define htau_hh2 (8.516)
#define ntau_hh2 (5.4586)

// Reversals:
#define ek (-77.00000f)
#define ena (50.00000f)

// Declarations:
__device__ void Curates_hh2(float v,float gnabar_hh2,float gkbar_hh2,float gl_hh2,float el_hh2);
float Cunernst(float ci,float co, float z) {
	if (z == 0) {
		return 0.;
	}
	if (ci <= 0.) {
		return 1e6;
	}else if (co <= 0.) {
		return -1e6;
	}else{
		return ktf/z*log(co/ci);
	}	
}

// Functions:
__device__ float Cuvtrap_hh2(float x,float y) {
        if (fabs(x/y) < 1e-6) {;
                return  y*(1 - x/y/2);
        }else{;
                return  x/(exp(x/y) - 1);
        };
};

// Procedures:
__device__ void Curates_hh2(float v,float gnabar_hh2,float gkbar_hh2,float gl_hh2,float el_hh2) {
float  alpha, beta, sum, q10;
                      ;
        q10 =pow((MYFTYPE) 3,(MYFTYPE)((celsius - 6.3)/10));
                ;
        alpha = .1 * Cuvtrap_hh2(-(v+40),10);
        beta =  4 * exp(-(v+65)/18);
        sum = alpha + beta;
/* removed mtau_hh2 recalculation */
       /* removed minf_hh2 recalculation */
                ;
        alpha = .07 * exp(-(v+65)/20);
        beta = 1 / (exp(-(v+35)/10) + 1);
        sum = alpha + beta;
/* removed htau_hh2 recalculation */
       /* removed hinf_hh2 recalculation */
                ;
        alpha = .01*Cuvtrap_hh2(-(v+55),10) ;
        beta = .125*exp(-(v+65)/80);
	sum = alpha + beta;
       /* removed ntau_hh2 recalculation */
       /* removed ninf_hh2 recalculation */
;};

// Inits:
__device__ void CuInitModel_hh2(float v,float &m,float &h,float &n,float gnabar_hh2,float gkbar_hh2,float gl_hh2,float el_hh2) {
float ;
	Curates_hh2(v,gnabar_hh2,gkbar_hh2,gl_hh2,el_hh2);
	m = minf_hh2;
	h = hinf_hh2;
	n = ninf_hh2;
;};

// Derivs:
__device__ void CuDerivModel_hh2(float dt, float v,float &m,float &h,float &n,float gnabar_hh2,float gkbar_hh2,float gl_hh2,float el_hh2) {
float ek,ik,il;
float ;
   Curates_hh2 (  v ,gnabar_hh2,gkbar_hh2,gl_hh2,el_hh2);
    m = m + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / mtau_hh2)))*(- ( ( ( minf_hh2 ) ) / mtau_hh2 ) / ( ( ( ( - 1.0) ) ) / mtau_hh2 ) - m) ;
    h = h + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / htau_hh2)))*(- ( ( ( hinf_hh2 ) ) / htau_hh2 ) / ( ( ( ( - 1.0) ) ) / htau_hh2 ) - h) ;
    n = n + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / ntau_hh2)))*(- ( ( ( ninf_hh2 ) ) / ntau_hh2 ) / ( ( ( ( - 1.0) ) ) / ntau_hh2 ) - n) ;
   ;}

// Breakpoints:
__device__ void CuBreakpointModel_hh2(MYSECONDFTYPE &sumCurrents, MYFTYPE &sumConductivity, float v,float &m,float &h,float &n,float gnabar_hh2,float gkbar_hh2,float gl_hh2,float el_hh2) {
float gk,gna;
float ik,il,ina;
gna=gnabar_hh2*m*m*m*h;
ina=gna*(v-ena);
gk=gkbar_hh2*n*n*n*n;
ik=gk*(v-ek);
il=gl_hh2*(v-el_hh2);
sumCurrents+= ik;
sumConductivity+= gk;
;};
