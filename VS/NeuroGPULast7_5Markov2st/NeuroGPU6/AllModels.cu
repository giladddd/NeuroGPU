#include "hip/hip_runtime.h"
// Automatically generated CU for C:\Users\rben.KECK-CENTER\Documents\GitHub\NeuroGPU\URapNeuron\Markov2st\runModel.hoc
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""
#include "AllModels.cuh"

// Universals:
#define PI (3.1415927f)
#define R (8.31441f)
#define FARADAY (96485.309f)
#define ktf (1000.*8.3134*(celsius + 273.15)/FARADAY)

#define _RHS1(arg) rhs[arg]
#define _MATELM1(i, j) matq[i][j]


// GGlobals
#define celsius (6.30000)
#define stoprun (0.00000)
#define clamp_resist (0.00100)
#define secondorder (0.00000)

// NGlobals:
#define _RHS1(arg) rhs[arg]
#define _MATELM1(i, j) matq[i][j]



// Reversals:
#define ek (-77.00000f)

// Declarations:
__device__ void Curates_CO(float v,float gbar_CO,float a12_CO,float a21_CO,float z12_CO,float z21_CO,float &k12,float &k21);
float Cunernst(float ci,float co, float z) {
	if (z == 0) {
		return 0.;
	}
	if (ci <= 0.) {
		return 1e6;
	}else if (co <= 0.) {
		return -1e6;
	}else{
		return ktf/z*log(co/ci);
	}	
}

// Kinetic Code:
__device__ void Cubackwards_euler( double h, int N, int nkinStates,float* rhs,float* y,float matq[2][2]){
  for (int i = 0; i < nkinStates; i++) {
        double w0 = y[i];
       for (int j = 0; j < N; j++) {
            double top = w0 - y[i] - h * rhs[i];
              double bottom = 1 - h * matq[i][i];
         double dw = top / bottom;
           w0 = w0 - dw;
             }
         y[i] = w0;
        }
     }

// Functions:

// Procedures:
__device__ void Curates_CO(float v,float gbar_CO,float a12_CO,float a21_CO,float z12_CO,float z21_CO,float &k12,float &k21) {
      k12 = a12_CO*exp(z12_CO*v);
      k21 = a21_CO*exp(-z21_CO*v);
     ;
;};

// Inits:
__device__ void CuInitModel_CO(float v,float &c1,float &o,float gbar_CO,float a12_CO,float a21_CO,float z12_CO,float z21_CO) {
float k12,k21;
double sum = 0;
        Curates_CO(v,gbar_CO,a12_CO,a21_CO,z12_CO,z21_CO,k12,k21);
//matq[0][1] =k12;
//matq[1][0] =k21;
for (int i = 0; i <2; i++) {
sum = 0 ;
for (int j = 0; j <2; j++) {
if (i != j) {
//sum +=matq[i][j];
;}
;}
//matq[i][i] = -sum;
;}
;}

// Derivs:
__device__ int CuDerivModel_CO(float dt, float v,float &c1,float &o,float gbar_CO,float a12_CO,float a21_CO,float z12_CO,float z21_CO) {
float k12,k21;
float rhs[2];
float y[2];
float matq[2][2];
y[0] =c1;
y[1] =o;
 {int _reset=0;
 {
   double b_flux, f_flux, _term; int _i;
 {int _i; double _dt1 = 1.0/dt;
for(_i=1;_i<2;_i++){
  	_RHS1(_i) = 0;
	_MATELM1(_i, _i) = _dt1;
      
;} ;}
 Curates_CO (  v ,gbar_CO,a12_CO,a21_CO,z12_CO,z21_CO,k12,k21);
   /* ~ c1 <-> o ( k12 , k21 )*/
 f_flux =  k12 * c1 ;
 b_flux =  k21 * o ;
 _RHS1( 1) -= (f_flux - b_flux);
 
 _term =  k12 ;
 _MATELM1( 1 ,1)  += _term;
 _term =  k21 ;
 _MATELM1( 1 ,0)  -= _term;
 /*REACTION*/
   /* c1 + o = 1.0 */
 _RHS1(0) =  1.0;
 _MATELM1(0, 0) = 1;
 _RHS1(0) -= o ;
 _MATELM1(0, 1) = 1;
 _RHS1(0) -= c1 ;
 Cubackwards_euler(dt,3,2,rhs,y,matq);
   ;} return _reset;
 ;}
 
;}

// Breakpoints:
__device__ void CuBreakpointModel_CO(MYSECONDFTYPE &sumCurrents, MYFTYPE &sumConductivity, float v,float &c1,float &o,float gbar_CO,float a12_CO,float a21_CO,float z12_CO,float z21_CO) {
float g,gk;
float ik;
g=gbar_CO*o;
ik=(1e-4)*g*(v-ek);
sumCurrents+= ik;
;};
