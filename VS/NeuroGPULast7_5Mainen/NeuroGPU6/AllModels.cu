#include "hip/hip_runtime.h"
// Automatically generated CU for C:\Users\bensr\Documents\GitHub\NeuroGPU\URapNeuron\Markov2st\runModel.hoc
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""
#include "AllModels.cuh"

// Universals:
#define PI (3.1415927f)
#define R (8.31441f)
#define FARADAY (96485.309f)
#define ktf (1000.*8.3134*(celsius + 273.15)/FARADAY)

// GGlobals
#define celsius (6.30000)
#define stoprun (0.00000)
#define clamp_resist (0.00100)
#define secondorder (0.00000)

// NGlobals:

// Reversals:
#define ek (-77.00000f)

// Declarations:
__device__ void Curates_CO(float v,float gbar_CO,float a12_CO,float a21_CO,float z12_CO,float z21_CO,float &k12,float &k21);
float Cunernst(float ci,float co, float z) {
	if (z == 0) {
		return 0.;
	}
	if (ci <= 0.) {
		return 1e6;
	}else if (co <= 0.) {
		return -1e6;
	}else{
		return ktf/z*log(co/ci);
	}	
}

// Functions:

// Procedures:
__device__ void Curates_CO(float v,float gbar_CO,float a12_CO,float a21_CO,float z12_CO,float z21_CO,float &k12,float &k21) {
      k12 = a12_CO*exp(z12_CO*v);
      k21 = a21_CO*exp(-z21_CO*v);
     ;
;};

// Inits:
__device__ void CuInitModel_CO(float v,float &c1,float &o,float gbar_CO,float a12_CO,float a21_CO,float z12_CO,float z21_CO) {

// Derivs:


// Breakpoints:
__device__ void CuBreakpointModel_CO(MYSECONDFTYPE &sumCurrents, MYFTYPE &sumConductivity, float v,float &c1,float &o,float gbar_CO,float a12_CO,float a21_CO,float z12_CO,float z21_CO) {
float g,gk;
float ik;
g=gbar_CO*o;
ik=(1e-4)*g*(v-ek);
sumCurrents+= ik;
;};
